#include "hip/hip_runtime.h"
﻿#include "on_collision.cuh"

#include <corecrt_math.h>

namespace fen
{
	__device__ void kernel_on_collision(const size_t o1, const size_t o2, const size_t& n, float const* positions, float const* radius, float* delta_mov, const float& dist_sq)
	{
		const float dist = sqrtf(dist_sq);
		const float min_dist = radius[o1] + radius[o2];

		const float col_axis_x = positions[o2] - positions[o1];
		const float col_axis_y = positions[o2 + n] - positions[o1 + n];

		const float mass_ratio_1 = radius[o1] / min_dist;
		const float mass_ratio_2 = radius[o2] / min_dist;

		const float delta = (dist - min_dist);

		const float n_x = col_axis_x / dist;
		const float n_y = col_axis_y / dist;

		delta_mov[o1] += n_x * mass_ratio_1 * delta;
		delta_mov[o1 + n] += n_y * mass_ratio_1 * delta;

		delta_mov[o2] -= n_x * mass_ratio_2 * delta;
		delta_mov[o2 + n] -= n_y * mass_ratio_2 * delta;
	}
}

