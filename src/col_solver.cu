#include "hip/hip_runtime.h"
﻿#include "col_solver.cuh"
#include "defines.h"
#include "on_collision.cuh"

#include <>
#include <hiprand.h>

#include <thrust/random.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>

#include <thrust/sort.h>
#include <thrust/remove.h>

INIT_INSTANCE_STATIC(fen::col_solver);

__global__ void kernel_scale(float* arr, const float scale, const float offset, const unsigned int n)
{
	for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
	{
		arr[i] = arr[i] * scale + offset;
	}
}

__device__ void kernel_sum_reduce(unsigned int* values, unsigned int* out)
{
	// wait for the whole array to be populated
	__syncthreads();

	// sum by reduction, using half the threads in each subsequent iteration
	unsigned int threads = blockDim.x;
	unsigned int half = threads / 2;

	while (half)
	{
		if (threadIdx.x < half)
		{
			// only keep going if the thread is in the first half threads
			for (int k = threadIdx.x + half; k < threads; k += half)
				values[threadIdx.x] += values[k];

			threads = half;
		}

		half /= 2;

		// make sure all the threads are on the same iteration
		__syncthreads();
	}

	// only let one thread update the current sum
	if (!threadIdx.x)
		atomicAdd(out, values[0]);
}

__global__ void kernel_init_cells(uint32_t* cells, uint32_t* objects, const float* positions, const float* radius, const float cell_dim,
	const float min_pos_x, const uint32_t max_cell_pos_x, const float min_pos_y, const uint32_t max_cell_pos_y, const size_t n, unsigned int* cell_count)
{
	const unsigned BITS = 16; // pos_x is allocated 15 bits because we need space for the home/phantom cell flag

	extern __shared__ unsigned int t[];
	unsigned int count = 0;

	for (unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
	{
		uint32_t hash = 0;
		unsigned int sides = 0;

		const int h = i * DIM_2;

		float dist;

		float x = positions[i] - min_pos_x;
		float y = positions[i + n] - min_pos_y;

		const uint32_t cell_pos_x = (uint32_t)(x / cell_dim);
		const uint32_t cell_pos_y = (uint32_t)(y / cell_dim);

		float rad = radius[i] * 1.41421356f; // sqrtf(2.0f)

		hash = cell_pos_x << BITS;
		hash = hash | cell_pos_y;

		const uint8_t home_cell_t = ((cell_pos_y & 0b1) << 1) | (cell_pos_x & 0b1);

		cells[h] = hash << 1 | 0b0;

		unsigned int home_cells_t_sides = 0b1 << home_cell_t;

		dist = y - floor(y / cell_dim) * cell_dim;
		if (dist < rad) // overlap with top cell
		{
			if (cell_pos_y > 0) // Not already in the top position
				sides |= 0x1;
		}
		else if (cell_dim - dist < rad)
		{
			if(cell_pos_y <= max_cell_pos_y) // overlap with bottom cell and not in the bottom position
				sides |= 0x2;
		}

		dist = x - floor(x / cell_dim) * cell_dim;
		sides <<= 2;
		if (dist < rad)
		{
			if (cell_pos_x > 0) // overlap with left cell
				sides |= 0x1;
		}
		else if (cell_dim - dist < rad) {
			if (cell_pos_x <= max_cell_pos_x)// overlap with right cell
				sides |= 0x2;
		}

		if (((sides >> 2) & 0x1) == 0x1) // check top
		{
			if ((sides & 0x1) == 0x1) // check left
			{
				// overlaps cells: top, top left, left
				cells[h + 1] = ((cell_pos_x << BITS) | (cell_pos_y - 1)) << 1 | 0b1;
				cells[h + 2] = (((cell_pos_x - 1) << BITS) | (cell_pos_y - 1)) << 1 | 0b1;
				cells[h + 3] = (((cell_pos_x - 1) << BITS) | cell_pos_y) << 1 | 0b1;

				home_cells_t_sides = 0b1111;

				count += 4;
			}
			else if ((sides & 0x2) == 0x2) // check right
			{
				// overlaps cells: top, top right, right
				cells[h + 1] = ((cell_pos_x << BITS) | (cell_pos_y - 1)) << 1 | 0b1;
				cells[h + 2] = (((cell_pos_x + 1) << BITS) | (cell_pos_y - 1)) << 1 | 0b1;
				cells[h + 3] = (((cell_pos_x + 1) << BITS) | cell_pos_y) << 1 | 0b1;

				home_cells_t_sides = 0b1111;

				count += 4;
			}
			else
			{
				// overlaps cells: top
				cells[h + 1] = ((cell_pos_x << BITS) | (cell_pos_y - 1)) << 1 | 0b1;

				home_cells_t_sides |= 0b1 << (home_cell_t + 2) % 4;

				count += 2;
			}
		}
		else if (((sides >> 2) & 0x2) == 0x2) // check bottom
		{
			if ((sides & 0x1) == 0x1) // check left
			{
				// overlaps cells: bottom, bottom left, left
				cells[h + 1] = ((cell_pos_x << BITS) | (cell_pos_y + 1)) << 1 | 0b1;
				cells[h + 2] = (((cell_pos_x - 1) << BITS) | (cell_pos_y + 1)) << 1 | 0b1;
				cells[h + 3] = (((cell_pos_x - 1) << BITS) | cell_pos_y) << 1 | 0b1;

				home_cells_t_sides = 0b1111;
				count += 4;
			}
			else if ((sides & 0x2) == 0x2) // check right
			{
				// overlaps cells: bottom, bottom right, right
				cells[h + 1] = ((cell_pos_x << BITS) | (cell_pos_y + 1)) << 1 | 0b1;
				cells[h + 2] = (((cell_pos_x + 1) << BITS) | (cell_pos_y + 1)) << 1 | 0b1;
				cells[h + 3] = (((cell_pos_x + 1) << BITS) | cell_pos_y) << 1 | 0b1;

				home_cells_t_sides = 0b1111;
				count += 4;
			}
			else
			{
				// overlaps cells: bottom
				cells[h + 1] = ((cell_pos_x << BITS) | (cell_pos_y + 1)) << 1 | 0b1;

				home_cells_t_sides |= 0b1 << (home_cell_t + 2) % 4;

				count += 2;
			}
		}
		else // check left and right
		{
			if ((sides & 0x1) == 0x1) // check left
			{
				// overlaps cells: left
				cells[h + 1] = (((cell_pos_x - 1) << BITS) | cell_pos_y) << 1 | 0b1;

				if (home_cell_t & 0b1)
					home_cells_t_sides |= 0b1 << (home_cell_t - 1);
				else
					home_cells_t_sides |= 0b1 << (home_cell_t + 1);

				count += 2;
			}
			else if ((sides & 0x2) == 0x2) // check right
			{
				// overlaps cells: right
				cells[h + 1] = (((cell_pos_x + 1) << BITS) | cell_pos_y) << 1 | 0b1;

				if (home_cell_t & 0b1)
					home_cells_t_sides |= 0b1 << (home_cell_t - 1);
				else
					home_cells_t_sides |= 0b1 << (home_cell_t + 1);

				count += 2;
			}
			else
			{
				// does not overlap with any other cell
				count++;
			}
		}

		objects[h] = (i << 7) | (home_cells_t_sides << 3) | (home_cell_t << 1) | 0b1;

		// Phantom cells
		objects[h + 1] = (i << 7) | (home_cells_t_sides << 3) | (home_cell_t << 1) | 0b0;
		objects[h + 2] = (i << 7) | (home_cells_t_sides << 3) | (home_cell_t << 1) | 0b0;
		objects[h + 3] = (i << 7) | (home_cells_t_sides << 3) | (home_cell_t << 1) | 0b0;
	}

	// perform reduction to count number of cells occupied
	t[threadIdx.x] = count;
	kernel_sum_reduce(t, cell_count);
}

__global__ void kernel_count_cols(uint32_t* cells, uint32_t* objects, float* positions, float* radius, unsigned int n, unsigned int m, unsigned int cells_per_thread, unsigned int* collision_count)
{
	extern __shared__ unsigned int t[];

	unsigned int thread_start = ((blockDim.x * blockIdx.x) + threadIdx.x) * cells_per_thread;

	if (thread_start >= m)
	{
		t[threadIdx.x] = 0;
		return;
	}

	unsigned int thread_end = thread_start + cells_per_thread;
	unsigned int i = thread_start;
	unsigned int cell;
	unsigned int collisions = 0;
	unsigned int h;
	unsigned int p;
	unsigned int start;
	unsigned int num_col_list;

	float d_c1, d_c2;
	uint32_t _c1, _c2;
	uint32_t t_c1, t_c2;
	uint32_t ts_c1, ts_c2;
	float dist, dx;

	if (thread_end > m)
	{
		thread_end = m;
	}

	// The first thread does not skip the first occurrence
	if (blockIdx.x == 0 && threadIdx.x == 0 || cells[thread_start - 1] >> 1 != cells[thread_start] >> 1)
		cell = UINT32_MAX;
	else
		cell = cells[thread_start] >> 1;

	while (true)
	{
		h = 0;
		p = 0;

		while (i < thread_end)
		{
			// Searches until it finds a valid home cell to start with 
			if ((cells[i] >> 1) == cell) //same as before or if it is a phantom cell
			{
				++i;
				continue;
			}

			// Found the first home cell
			cell = cells[i] >> 1;
			start = i;
			break;
		}

		// If i reached the end AND the end is not the start of a new collision list
		if (i >= thread_end)
			break;

		while ((cells[i] >> 1) == cell)
		{
			if (objects[i] & 0x01)
				++h;
			else
				++p;
			++i;
		}

		num_col_list = h + p;

		if (h > 0 && num_col_list > 1) {

			for (unsigned int c1 = 0; c1 < h; ++c1)
			{
				unsigned int offset = start + c1;
				_c1 = objects[offset] >> 7;
				t_c1 = objects[offset] >> 1 & 0b11;
				ts_c1 = objects[offset] >> 3 & 0b1111;

				d_c1 = radius[_c1];

				for (unsigned int c2 = c1 + 1; c2 < num_col_list; ++c2)
				{
					offset = start + c2;
					_c2 = objects[offset] >> 7;
					t_c2 = objects[offset] >> 1 & 0b11;
					ts_c2 = objects[offset] >> 3 & 0b1111;

					d_c2 = radius[_c2];

					dist = 0;

					if (t_c2 < t_c1 && (0b1 << t_c2 & ts_c1) && (0b1 << t_c1 & ts_c2))
						continue;

					for (int l = 0; l < DIM; ++l)
					{
						dx = positions[_c2 + l * n] - positions[_c1 + l * n];
						dist += dx * dx;
					}

					if (dist < ((d_c1 + d_c2) * (d_c2 + d_c1)))
					{
						collisions++;
					}
				}
			}
		}
	}

	t[threadIdx.x] = collisions;

	__syncthreads();

	if(!threadIdx.x)
	{
		atomicAdd(collision_count, thrust::reduce(thrust::device, t, t + blockDim.x));
	}
}

__global__ void kernel_check_cell_cols(uint32_t* cells, uint32_t* objects, unsigned int m, unsigned int cells_per_thread, uint64_t* col_cells, unsigned int* error_flag)
{
	unsigned int thread_start = ((blockDim.x * blockIdx.x) + threadIdx.x) * cells_per_thread;

	if (thread_start >= m)
	{
		return;
	}

	unsigned int thread_end = thread_start + cells_per_thread;
	unsigned int i = thread_start;
	unsigned int cell;
	uint64_t h;
	uint64_t p;
	uint64_t start = thread_start;
	unsigned int num_col_list;

	if (thread_end > m)
	{
		thread_end = m;
	}

	// The first thread does not skip the first occurrence
	if (blockIdx.x == 0 && threadIdx.x == 0 || cells[thread_start - 1] >> 1 != cells[thread_start] >> 1)
		cell = UINT32_MAX;
	else
		cell = cells[thread_start] >> 1;

	while (i < m)
	{
		h = 0;
		p = 0;

		while (i < thread_end)
		{
			// Searches until it finds a valid home cell to start with 
			if ((cells[i] >> 1) == cell) //same as before or if it is a phantom cell
			{
				++i;
				continue;
			}

			// Found the first home cell
			cell = cells[i] >> 1;
			start = i;
			break;
		}

		// If i reached the end AND the end is not the start of a new collision list
		if (i >= thread_end)
			break;

		while ((cells[i] >> 1) == cell)
		{
			if (objects[i] & 0x01)
				++h;
			else
				++p;
			++i;
		}

		num_col_list = h + p;

		// A collision cell
		if (h > 0 && num_col_list > 1) {
			if (start < START_LIMIT && h < HOME_LIMIT && p < PHANTOM_LIMIT)
			{
				col_cells[start] = (start << BITS_OFFSET_START) | (h << BITS_OFFSET_HOME) | p;
			}
			else
				*error_flag = true;
		}
	}
}

__global__ void kernel_solve_cols(const uint64_t* col_cells, const uint32_t* objects, const float* positions, const float* radius, float* delta_mov, const uint32_t n, const unsigned int m, const unsigned char cell_type, unsigned int* collision_count)
{
	extern __shared__ unsigned int t[];

	unsigned int collisions = 0;

	for (unsigned int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < m; i += (gridDim.x * blockDim.x))
	{
		const uint64_t& col_cell_data = col_cells[i];

		const unsigned int p = col_cell_data & (PHANTOM_LIMIT - 1u); 
		const unsigned int h = (col_cell_data >> BITS_OFFSET_HOME) & (HOME_LIMIT - 1u); 
		const unsigned int start = (col_cell_data >> BITS_OFFSET_START) & (START_LIMIT - 1u); 

		if(cell_type == ((objects[start] >> 1) & 0b11))
		{
			float d_c1, d_c2;
			uint32_t _c1, _c2;
			uint32_t t_c1, t_c2;
			uint32_t ts_c1, ts_c2;
			float dist, dx;

			const unsigned int num_col_list = h + p;

			for (unsigned int c1 = 0; c1 < h; ++c1)
			{
				unsigned int offset = start + c1;
				_c1 = objects[offset] >> 7;
				t_c1 = objects[offset] >> 1 & 0b11;
				ts_c1 = objects[offset] >> 3 & 0b1111;

				d_c1 = radius[_c1];

				for (unsigned int c2 = c1 + 1; c2 < num_col_list; ++c2)
				{
					offset = start + c2;
					_c2 = objects[offset] >> 7;
					t_c2 = objects[offset] >> 1 & 0b11;
					ts_c2 = objects[offset] >> 3 & 0b1111;

					d_c2 = radius[_c2] + d_c1;

					dist = 0;

					if (t_c2 < t_c1 && (0b1 << t_c2 & ts_c1) && (0b1 << t_c1 & ts_c2))
						continue;

					for (int l = 0; l < DIM; ++l)
					{
						dx = positions[_c2 + l * n] - positions[_c1 + l * n];
						dist += dx * dx;
					}

					if (dist < (d_c2 * d_c2))
					{
						collisions++;
						kernel_on_collision(_c1, _c2, n, positions, radius, delta_mov, dist);
					}
				}
			}
		}
	}

	t[threadIdx.x] = collisions;

	__syncthreads();

	if (!threadIdx.x)
	{
		atomicAdd(collision_count, thrust::reduce(thrust::device, t, t + blockDim.x));
	}
}

__global__ void kernel_move_entities(float* positions, float* delta_mov, const uint32_t n, const float min_pos_x, const float max_pos_x, const float min_pos_y, const float max_pos_y)
{
	for (unsigned int i = (blockDim.x * blockIdx.x) + threadIdx.x; i < n; i += (gridDim.x * blockDim.x))
	{
		positions[i] += delta_mov[i];
		positions[i + n] += delta_mov[i + n];

		if (positions[i] < min_pos_x) positions[i] = min_pos_x;
		else if (positions[i] > max_pos_x) positions[i] = max_pos_x;

		if (positions[i + n] < min_pos_y) positions[i + n] = min_pos_y;
		else if (positions[i + n] > max_pos_y) positions[i + n] = max_pos_y;
	}
}


#define GET_RAW_PTR(v) thrust::raw_pointer_cast(v.data())

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace fen
{

void init_objects(unsigned long long seed, unsigned num_blocks, unsigned num_threads, thrust::device_vector<float>& positions, thrust::device_vector<float>& radius, thrust::device_vector<float>& delta_mov, float min_pos_x, float max_pos_x, float min_pos_y, float max_pos_y, float min_radius, float max_radius)
{
	auto& generator = col_solver::Instance()->get_generator();
	hiprandSetPseudoRandomGeneratorSeed(generator, seed);
	
	hiprandGenerateUniform(generator, GET_RAW_PTR(positions), positions.size());
	CUDA_CALL_2(kernel_scale, num_blocks, num_threads)(GET_RAW_PTR(positions), max_pos_x - min_pos_x, min_pos_x, positions.size() / 2);
	CUDA_CALL_2(kernel_scale, num_blocks, num_threads)(GET_RAW_PTR(positions) + positions.size() / 2, max_pos_y - min_pos_y, min_pos_y, positions.size() / 2);

	hiprandGenerateUniform(generator, GET_RAW_PTR(radius), radius.size());
	CUDA_CALL_2(kernel_scale, num_blocks, num_threads)(GET_RAW_PTR(radius), max_radius - min_radius, min_radius, radius.size());

	thrust::fill(thrust::device, delta_mov.begin(), delta_mov.end(), 0.0f);
}


unsigned int col_solver::solve_cols_1(unsigned num_blocks, unsigned num_threads, thrust::device_vector<float>& positions, thrust::device_vector<float>& radius, thrust::device_vector<float>& delta_mov, const size_t num_entities)
{

	profiler.start_timing<Cells_Init>();
	unsigned int num_cells = init_cells(num_blocks, num_threads, positions, radius, num_entities);
	hipDeviceSynchronize();
	profiler.finish_timing<Cells_Init>();

	profiler.start_timing<Sort>();
	sort_cells();
	hipDeviceSynchronize();
	profiler.finish_timing<Sort>();

	unsigned int collisions = count_cols_1(num_blocks, num_threads, positions, radius, delta_mov, num_entities, num_cells);

	profiler.start_timing<Move>();
	CUDA_CALL_2(kernel_move_entities, num_blocks, num_threads)(GET_RAW_PTR(positions), GET_RAW_PTR(delta_mov), num_entities, min_pos_x, max_pos_x, min_pos_y, max_pos_y);
	thrust::fill(thrust::device, delta_mov.begin(), delta_mov.end(), 0.0f);
	hipDeviceSynchronize();
	profiler.finish_timing<Move>();

	return collisions;
}

col_solver::col_solver() : Singleton()
{
	hipMalloc((void**)&temp, sizeof(unsigned int));

	hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
}

col_solver::~col_solver()
{
	hipFree(temp);
}

void col_solver::init_solver(const size_t num_entities, const float max_rad_, const float min_pos_x_, const float min_pos_y_, const float max_pos_x_, const float max_pos_y_)
{
	//printf("Init solver\n");
	cells = thrust::device_vector<uint32_t>(num_entities * DIM_2);
	objects = thrust::device_vector<uint32_t>(num_entities * DIM_2);
	col_cells = thrust::device_vector<uint64_t>();

	max_rad = max_rad_;

	cell_size = max_rad * 4.0f;

	min_pos_x = min_pos_x_;
	min_pos_y = min_pos_y_;
	max_pos_x = max_pos_x_;
	max_pos_y = max_pos_y_;

	width = max_pos_x_ - min_pos_x_;
	height = max_pos_y_ - min_pos_y_;

	profiler.reset();
}

unsigned int col_solver::init_cells(unsigned num_blocks, unsigned num_threads, thrust::device_vector<float>& positions, thrust::device_vector<float>& radius, const size_t num_entities)
{
	//printf("Init ");

	// reset
	hipMemset(GET_RAW_PTR(cells), 0xff, num_entities * DIM_2 * sizeof(decltype(cells)::value_type));
	hipMemset(temp, 0, sizeof(unsigned int));

	// If max rad isn't specified, choose from largest radius
	if (max_rad < 0)
		cell_size = *thrust::max_element(thrust::device, radius.cbegin(), radius.cend());

	CUDA_CALL_3(kernel_init_cells, num_blocks, num_threads, num_threads * sizeof(unsigned int))(GET_RAW_PTR(cells), GET_RAW_PTR(objects), GET_RAW_PTR(positions), GET_RAW_PTR(radius),
		cell_size, min_pos_x, (uint32_t)(width / cell_size), min_pos_y, (uint32_t)(height / cell_size), num_entities, temp);

	gpuErrchk(hipPeekAtLastError());

	unsigned int num_cells = 0;
	hipMemcpy(&num_cells, temp, sizeof(unsigned int), hipMemcpyDeviceToHost);

	//printf(" n_cells:%u ", num_cells);
	return num_cells;
}

void col_solver::sort_cells()
{
	//printf("Sort ");

	thrust::stable_sort_by_key(thrust::device, cells.begin(), cells.end(), objects.begin(), thrust::less<uint32_t>());
}

unsigned int col_solver::count_cols_1(unsigned num_blocks, unsigned num_threads, thrust::device_vector<float>& positions, thrust::device_vector<float>& radius, thrust::device_vector<float>& delta_mov, const size_t num_entities, const unsigned num_cells)
{
	//printf("Count\n");

	profiler.start_timing<Cols_Init>();

	unsigned int cells_per_thread = ((num_cells - 1) / num_blocks) /
		num_threads +
		1;

	col_cells.resize(num_cells);

	// Reset to 0
	hipMemset(GET_RAW_PTR(col_cells), 0x00, col_cells.size() * sizeof(decltype(col_cells)::value_type));
	hipMemset(temp, 0, sizeof(unsigned int));

	CUDA_CALL_2(kernel_check_cell_cols, num_blocks, num_threads) (
		GET_RAW_PTR(cells), GET_RAW_PTR(objects),
		num_cells,
		cells_per_thread,
		GET_RAW_PTR(col_cells),
		temp
	);

	gpuErrchk(hipPeekAtLastError());

	unsigned int error_flag = 0;
	hipMemcpy(&error_flag, temp, sizeof(unsigned int), hipMemcpyDeviceToHost);

	if(error_flag)
	{
		printf("Error: Too many entities in a single cell\n");
		return 0;
	}

	const unsigned int dist = thrust::remove(col_cells.begin(), col_cells.end(), 0u) - col_cells.begin();

	num_blocks = std::min<unsigned int>(num_blocks, (dist / num_threads) + 1u);


	profiler.finish_timing<Cols_Init>();

	profiler.start_timing<Cols_Resolve>();

	for (int i = 0; i < 4; ++i)
	{
		CUDA_CALL_3(kernel_solve_cols, num_blocks, num_threads, num_threads * sizeof(unsigned int)) (
			GET_RAW_PTR(col_cells), GET_RAW_PTR(objects),
			GET_RAW_PTR(positions), GET_RAW_PTR(radius), GET_RAW_PTR(delta_mov),
			num_entities,
			dist,
			i,
			temp
		);

		gpuErrchk(hipPeekAtLastError());
	}

	hipDeviceSynchronize();
	gpuErrchk(hipPeekAtLastError());

	unsigned int collisions = 0;
	hipMemcpy(&collisions, temp, sizeof(unsigned int), hipMemcpyDeviceToHost);

	profiler.finish_timing<Cols_Resolve>();

	return collisions;
}

}
