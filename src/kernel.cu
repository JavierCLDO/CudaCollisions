﻿#include "col_solver.cuh"
#include "defines.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <chrono>
#include <future>

#include "simple_profiler.h"

unsigned int test_cpu_cols(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_objects)
{
	unsigned int collisions = 0;
	float dh;
	float dp;
	float dx;
	float d;

	for (unsigned int j = 0; j < num_objects; j++) {
		dh = radius[j];

		for (unsigned int k = j + 1; k < num_objects; k++) {

			// assume dims are radii of balls
			dp = radius[k];

			d = 0;

			for (unsigned int l = 0; l < DIM; l++) {
				dx = positions[j + l * num_objects] - positions[k + l * num_objects];
				d += dx * dx;
			}

			auto d_s = (dh + dp) * (dp + dh);

			// if collision
			if (d < d_s) {
				collisions++;
			}
		}
	}

	return collisions;
}

void print_entities(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_entities, unsigned int n)
{
	for (int i = 1020; i < 1020 + n; i++) {

		printf("\n%.4i: ", i);

		for (int j = 0; j < DIM; j++) {
			printf(" %10.5f ", positions[i + j * num_entities]);
		}

		printf("\t%.5f", radius[i]);
	}

	printf("\n");
}

float rand_float(const float& low, const float& high)
{
	return low + static_cast<float>(rand()) * (high - low) / RAND_MAX;
}

int main()
{
	fen::col_solver::CreateInstance();

	constexpr size_t NUM_OBJECTS = 1 << 18;

	constexpr unsigned int NUM_BLOCKS = 100;
	constexpr unsigned int NUM_THREADS = 512;

	constexpr float MIN_RAD = 0.5f;
	constexpr float MAX_RAD = 1.0f;

	constexpr float MIN_X = -512.0f;
	constexpr float MIN_Y = -512.0f;
	constexpr float MAX_X = +512.0f;
	constexpr float MAX_Y = +512.0f;

	auto seed = time(nullptr);
	srand(seed);

	printf("Seed: %llu\n", seed);


	printf("MAX_X: %.2f; MAX_Y: %.2f\t\n", MAX_X, MAX_Y);

	constexpr bool CALCULATE_RAD = false;

	auto positions = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto delta_mov = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto radius = thrust::device_vector<float>(NUM_OBJECTS);

	auto col_solver = fen::col_solver::Instance();

	col_solver->init_objects(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, MIN_X, MAX_X, MIN_Y, MAX_Y, MIN_RAD, MAX_RAD);

	thrust::host_vector<float> h_positions = positions;
	thrust::host_vector<float> h_radius = radius;

	//print_entities(h_positions, h_radius, NUM_OBJECTS, 16);

	col_solver->reset(NUM_OBJECTS, CALCULATE_RAD ? -1.0f : MAX_RAD, MIN_X, MIN_Y, MAX_X, MAX_Y);

	for (int i = 0; i < 30; ++i) {

		const unsigned int collisions_1 = col_solver->solve_cols_1(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		for (int s = 0; s < 8 - 1; ++s)
		{
			col_solver->solve_cols_1(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		}

		col_solver->get_profiler().next_step();

		printf("col:%u\n", collisions_1);
	}

	col_solver->get_profiler().print_avg_times(std::cout);
	col_solver->get_profiler().print_times(std::cout);

    return 0;
}
