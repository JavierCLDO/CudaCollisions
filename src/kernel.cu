#include "hip/hip_runtime.h"
﻿#include "col_solver.cuh"
#include "defines.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <chrono>
#include <future>
#include <thread>

unsigned int test_cpu_cols(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_objects)
{
	unsigned int collisions = 0;
	float dh;
	float dp;
	float dx;
	float d;

	for (unsigned int j = 0; j < num_objects; j++) {
		dh = radius[j];

		for (unsigned int k = j + 1; k < num_objects; k++) {

			// assume dims are radii of balls
			dp = radius[k];

			d = 0;

			for (unsigned int l = 0; l < DIM; l++) {
				dx = positions[j + l * num_objects] - positions[k + l * num_objects];
				d += dx * dx;
			}

			auto d_s = (dh + dp) * (dp + dh);

			// if collision
			if (d < d_s) {
				collisions++;
			}
		}
	}

	return collisions;
}

void print_entities(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_entities, unsigned int n)
{
	for (int i = 1020; i < 1020 + n; i++) {

		printf("\n%.4i: ", i);

		for (int j = 0; j < DIM; j++) {
			printf(" %10.5f ", positions[i + j * num_entities]);
		}

		printf("\t%.5f\n", radius[i]);
	}

	printf("\n");
}

float rand_float(const float& low, const float& high)
{
	return low + static_cast<float>(rand()) * (high - low) / RAND_MAX;
}

void test_col(float pos1_x, float pos1_y, float pos2_x, float pos2_y, float radius1, float radius2, float& delta_mov1_x, float& delta_mov1_y, float& delta_mov2_x, float& delta_mov2_y)
{
	delta_mov1_x = delta_mov1_y = delta_mov2_x = delta_mov2_y = 0.0f;

	const float col_axis_x = pos2_x - pos1_x;
	const float col_axis_y = pos2_y - pos1_y;

	const float dist_sq = (col_axis_x * col_axis_x) + (col_axis_y * col_axis_y);

	const float dist = sqrtf(dist_sq);

	const float min_dist = radius1 + radius2;

	if (dist >= min_dist)
		return;

	const float mass_ratio_1 = radius1 / min_dist;
	const float mass_ratio_2 = radius2 / min_dist;

	const float delta = (dist - min_dist);

	const float n_x = col_axis_x / dist;
	const float n_y = col_axis_y / dist;

	printf("c_dist: %.3f; ", dist);

	delta_mov1_x += n_x * mass_ratio_1 * delta;
	delta_mov1_y += n_y * mass_ratio_1 * delta;

	delta_mov2_x -= n_x * mass_ratio_2 * delta;
	delta_mov2_y -= n_y * mass_ratio_2 * delta;



}

int main()
{
	printf("\n CPU: \n");
	float pos1_x = 10.0f, pos1_y = 10.0f, pos2_x = 11.0f, pos2_y = 11.0f;
	float delta_mov1_x = 0.0f, delta_mov1_y = 0.0f, delta_mov2_x = 0.0f, delta_mov2_y = 0.0f;

	test_col(pos1_x, pos1_y, pos2_x, pos2_y, 
		0.9f, 0.9f, delta_mov1_x, delta_mov1_y, delta_mov2_x, delta_mov2_y);

	test_col(pos1_x + delta_mov1_x, pos1_y + delta_mov1_y, pos2_x + delta_mov2_x, pos2_y + delta_mov2_x, 
		0.9f, 0.9f, delta_mov1_x, delta_mov1_y, delta_mov2_x, delta_mov2_y);

	printf("\n GPU: \n");
	constexpr size_t NUM_OBJECTS = 1 << 18;

	constexpr unsigned int NUM_BLOCKS = 100;
	constexpr unsigned int NUM_THREADS = 512;

	constexpr float   = 0.5f;
	constexpr float MAX_RAD = 1.0f;

	constexpr float MIN_X = 4.f;
	constexpr float MIN_Y = 4.f;

	fen::col_solver::Create();

	auto seed = time(nullptr);
	srand(seed);

	printf("Seed: %llu\n", seed);

	float MAX_X = 1024.0f;
	float MAX_Y = 1024.0f;

	printf("MAX_X: %.2f; MAX_Y: %.2f\t\n", MAX_X, MAX_Y);

	constexpr bool CALCULATE_RAD = false;

	auto positions = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto delta_mov = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto radius = thrust::device_vector<float>(NUM_OBJECTS);

	//fen::hack_init(NUM_BLOCKS, 1024, positions, radius, MIN_RAD, MAX_RAD);
	fen::col_solver::Instance()->init_objects(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, MIN_X, MAX_X - MIN_X, MIN_Y, MAX_Y - MIN_Y, MIN_RAD, MAX_RAD);

	thrust::host_vector<float> h_positions = positions;
	thrust::host_vector<float> h_radius = radius;

	//print_entities(h_positions, h_radius, NUM_OBJECTS, 16);

	fen::col_solver::Instance()->reset(NUM_OBJECTS, CALCULATE_RAD ? -1.0f : MAX_RAD, MIN_X, MIN_Y, MAX_X, MAX_Y);

	for (int i = 0; i < 60; ++i) {
		const auto start = std::chrono::high_resolution_clock::now();

		unsigned int collisions_1 = fen::col_solver::Instance()->solve_cols_1(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		for (int s = 0; s < 8 - 1; ++s)
		{
			fen::col_solver::Instance()->solve_cols_1(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		}

		printf("col:%u", collisions_1);

		const auto end = std::chrono::high_resolution_clock::now();
		const std::chrono::duration<double, std::milli> ms_double = end - start;
		printf("\tElapsed time: %.3f ms\n", ms_double);

	}

    return 0;
}
