﻿#include "col_solver.cuh"
#include "defines.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <future>
#include <cstdlib>
#include <sstream>

#include "simple_profiler.h"

unsigned int test_cpu_cols(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_objects)
{
	unsigned int collisions = 0;
	float dh;
	float dp;
	float dx;
	float d;

	for (unsigned int j = 0; j < num_objects; j++) {
		dh = radius[j];

		for (unsigned int k = j + 1; k < num_objects; k++) {

			// assume dims are radii of balls
			dp = radius[k];

			d = 0;

			for (unsigned int l = 0; l < DIM; l++) {
				dx = positions[j + l * num_objects] - positions[k + l * num_objects];
				d += dx * dx;
			}

			auto d_s = (dh + dp) * (dp + dh);

			// if collision
			if (d < d_s) {
				collisions++;
			}
		}
	}

	return collisions;
}

void print_entities(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_entities, unsigned int n)
{
	for (unsigned i = 1020u; i < 1020u + n; i++) {

		printf("\n%.4i: ", i);

		for (int j = 0; j < DIM; j++) {
			printf(" %10.5f ", positions[i + j * num_entities]);
		}

		printf("\t%.5f", radius[i]);
	}

	printf("\n");
}

float rand_float(const float& low, const float& high)
{
	return low + static_cast<float>(rand()) * (high - low) / RAND_MAX;
}

void print_usage(const char* name)
{
	printf("defaults args are: 30 18 100 512 8 0.5 1.0 -2048.0 -2048.0 2048.0 2048.0\n");

	printf("usage: \t%s \n"
		"\t%s help\n"
		"\t%s ITERATIONS\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n)\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n) NUM_BLOCKS\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n) NUM_BLOCKS NUM_THREADS\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n) NUM_BLOCKS NUM_THREADS SUB_STEPS\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n) NUM_BLOCKS NUM_THREADS SUB_STEPS RAD\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n) NUM_BLOCKS NUM_THREADS SUB_STEPS MIN_RAD MAX_RAD\n"
		"\t%s ITERATIONS NUM_OBJECTS(2^n) NUM_BLOCKS NUM_THREADS SUB_STEPS MIN_RAD MAX_RAD MIN_X MIN_Y MAX_X MAX_Y \n"
		, name, name, name, name, name, name, name, name, name, name
	);
}

template<typename T>
T get_var(const char* arg)
{

	std::istringstream iss(arg);

	T var;
	if (iss >> var) 
		return var;

	throw std::invalid_argument("wrong argument");
}

int main(int argc, char** argv)
{
	fen::col_solver::CreateInstance();

	size_t NUM_OBJECTS = 1 << 18;

	unsigned int NUM_BLOCKS = 100;
	unsigned int NUM_THREADS = 512;
	unsigned int SUB_STEPS = 8;
	unsigned int ITERATIONS = 30;
	float MIN_RAD = 0.5f;
	float MAX_RAD = 1.0f;

	float MIN_X = -2048.0f;
	float MIN_Y = -2048.0f;
	float MAX_X = +2048.0f;
	float MAX_Y = +2048.0f;

	int it = 1;

	try
	{
		if (argc > it) {

			if(!strcmp("help", argv[it]))
			{
				print_usage(argv[0]);
				return 0;
			}

			ITERATIONS = get_var<unsigned>(argv[it++]);
		}
		if (argc > it) NUM_OBJECTS = 1 << get_var<unsigned>(argv[it++]);
		if (argc > it) NUM_BLOCKS = get_var<unsigned>(argv[it++]);
		if (argc > it) NUM_THREADS = get_var<unsigned>(argv[it++]);
		if (argc > it) SUB_STEPS = get_var<unsigned>(argv[it++]);
		if (argc > it) MIN_RAD = MAX_RAD = get_var<float>(argv[it++]);
		if (argc > it) MAX_RAD = get_var<float>(argv[it++]);
		if (argc > it + 3) {
			MIN_X = get_var<float>(argv[it++]);
			MIN_Y = get_var<float>(argv[it++]);
			MAX_X = get_var<float>(argv[it++]);
			MAX_Y = get_var<float>(argv[it++]);
		}
		if (MAX_RAD < MIN_RAD ||
			MIN_X + MAX_RAD >= MAX_X ||
			MIN_Y + MAX_RAD >= MAX_Y ||
			MAX_X - MAX_RAD <= MIN_X ||
			MAX_Y - MAX_RAD <= MIN_Y)
		{
			print_usage(argv[0]);
			return 0;
		}
	} catch (...)
	{
		print_usage(argv[0]);
		return 0;
	}


	auto seed = time(nullptr);
	srand(seed);

	printf("Seed: %llu\n", seed);

	printf("Args used: ITERATIONS: %u; NUM_OBJECTS: %u; NUM_BLOCKS: %u; NUM_THREADS: %u; SUB_STEPS: %u;\n\tMIN_RAD: %.2f; MAX_RAD: %.2f; MIN_X: %.2f; MIN_Y: %.2f; MAX_X: %.2f; MAX_Y: %.2f\n", 
		ITERATIONS, NUM_OBJECTS, NUM_BLOCKS, NUM_THREADS, SUB_STEPS, MIN_RAD, MAX_RAD, MIN_X, MIN_Y, MAX_X, MAX_Y);

	constexpr bool CALCULATE_RAD = false;

	auto positions = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto delta_mov = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto radius = thrust::device_vector<float>(NUM_OBJECTS);

	auto col_solver = fen::col_solver::Instance();

	fen::init_objects(seed, NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, MIN_X, MAX_X, MIN_Y, MAX_Y, MIN_RAD, MAX_RAD);

	thrust::host_vector<float> h_positions = positions;
	thrust::host_vector<float> h_radius = radius;

	//print_entities(h_positions, h_radius, NUM_OBJECTS, 16);

	col_solver->reset(NUM_OBJECTS, CALCULATE_RAD ? -1.0f : MAX_RAD, MIN_X, MIN_Y, MAX_X, MAX_Y);

	for (int i = 0; i < ITERATIONS; ++i) {

		const unsigned int collisions_1 = col_solver->solve_cols_1(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		for (unsigned s = 0; s < SUB_STEPS - 1u; ++s)
		{
			col_solver->solve_cols_1(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		}

		col_solver->get_profiler().next_step();
		printf("col:%u\n", collisions_1);
	}
	std::cout << '\n';
	col_solver->get_profiler().print_avg_times<Solver_Execution_Steps>(std::cout);
	std::cout << '\n';
	col_solver->get_profiler().print_times(std::cout);

    return 0;
}
