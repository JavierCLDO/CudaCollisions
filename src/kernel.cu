﻿#include "col_solver.cuh"
#include "defines.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <future>
#include <cstdlib>
#include <sstream>

#include "simple_profiler.h"

/**
 * \brief Used to test whether the results from col_solver were correct or not
 * \param positions host vector
 * \param radius host vector
 * \param num_objects number of entities
 * \return collisions
 */
unsigned int test_cpu_cols(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_objects)
{
	unsigned int collisions = 0;
	float dh;
	float dp;
	float dx;
	float d;

	for (unsigned int j = 0; j < num_objects; j++) {
		dh = radius[j];

		for (unsigned int k = j + 1; k < num_objects; k++) {

			// assume dims are radii of balls
			dp = radius[k];

			d = 0;

			for (unsigned int l = 0; l < DIM; l++) {
				dx = positions[j + l * num_objects] - positions[k + l * num_objects];
				d += dx * dx;
			}

			auto d_s = (dh + dp) * (dp + dh);

			// if collision
			if (d < d_s) {
				collisions++;
			}
		}
	}

	return collisions;
}


/**
 * \brief Prints n random entities
 * \param positions host vector
 * \param radius host vector
 * \param num_entities number of entities
 * \param n number of prints to do
 */
void print_entities(thrust::host_vector<float>& positions, thrust::host_vector<float>& radius, size_t num_entities, unsigned int n)
{
	for (unsigned i = 0u; i < n; i++) {

		int e = rand() % n;

		printf("\n%.4i: ", i);

		for (int j = 0; j < DIM; j++) {
			printf(" %10.5f ", positions[i + j * num_entities]);
		}

		printf("\t%.5f", radius[i]);
	}

	printf("\n");
}


/**
 * \brief Prints usage
 * \param name executable file name
 */
void print_usage(const char* name)
{
	printf("defaults args are: 30 18 100 512 8 0.5 1.0 0 -2048.0 -2048.0 2048.0 2048.0\n");

	printf("usage: \t%s \n", name);
	printf("\t%s help\n", name);
	printf("\t%s ITERATIONS\n", name);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu)\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS NUM_THREADS(<=1024)\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS NUM_THREADS(<=1024) SUB_STEPS\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS NUM_THREADS(<=1024) SUB_STEPS RAD\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS NUM_THREADS(<=1024) SUB_STEPS MIN_RAD MAX_RAD\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS NUM_THREADS(<=1024) SUB_STEPS MIN_RAD MAX_RAD CALCULATE_RAD\n", name, BITS_START - 2);
	printf("\t%s ITERATIONS NUM_OBJECTS(2^n, n<%llu) NUM_BLOCKS NUM_THREADS(<=1024) SUB_STEPS MIN_RAD MAX_RAD CALCULATE_RAD MIN_X MIN_Y MAX_X MAX_Y\n", name, BITS_START - 2);
}


/**
 * \brief Gets a var value from c string
 * \tparam T Type of value to read
 * \param arg c string
 * \return the value
 * \throws invalid_argument: when the parsing fails (due to mismatch of types)
 */
template<typename T>
T get_var(const char* arg)
{

	std::istringstream iss(arg);

	T var;
	if (iss >> var) 
		return var;

	std::ostringstream ss;
	ss << "wrong argument error -> " << typeid(T).name() << ": " << arg;
	throw std::invalid_argument(ss.str());
}


/**
 * \brief Reads and applies command line arguments
 * \return False when a wrong argument was inputted
 */
bool read_args(int argc, char** argv, size_t& NUM_OBJECTS, unsigned int& NUM_BLOCKS, unsigned int& NUM_THREADS, unsigned int& SUB_STEPS, unsigned int& ITERATIONS, float& MIN_RAD, float& MAX_RAD, bool& CALCULATE_RAD, float& MIN_X, float& MIN_Y, float& MAX_X, float& MAX_Y)
{
	int it = 1;
	unsigned int n_objects_raise = 0;

	try
	{
		if (argc > it) {

			if (!strcmp("help", argv[it]))
			{
				return false;
			}

			ITERATIONS = get_var<unsigned>(argv[it++]);
		}
		if (argc > it) n_objects_raise = get_var<unsigned>(argv[it++]);
		if (argc > it) NUM_BLOCKS = get_var<unsigned>(argv[it++]);
		if (argc > it) NUM_THREADS = get_var<unsigned>(argv[it++]);
		if (argc > it) SUB_STEPS = get_var<unsigned>(argv[it++]);
		if (argc > it) MIN_RAD = MAX_RAD = get_var<float>(argv[it++]);
		if (argc > it) MAX_RAD = get_var<float>(argv[it++]);
		if (argc > it) CALCULATE_RAD = get_var<bool>(argv[it++]);
		if (argc > it + 3) {
			MIN_X = get_var<float>(argv[it++]);
			MIN_Y = get_var<float>(argv[it++]);
			MAX_X = get_var<float>(argv[it++]);
			MAX_Y = get_var<float>(argv[it++]);
		}

		// Ranges check
		if (n_objects_raise > (BITS_START - 2) ||
			NUM_THREADS > 1024 || 
			MAX_RAD < MIN_RAD ||
			MIN_X + MAX_RAD >= MAX_X ||
			MIN_Y + MAX_RAD >= MAX_Y ||
			MAX_X - MAX_RAD <= MIN_X ||
			MAX_Y - MAX_RAD <= MIN_Y)
		{
			return false;
		}
	}
	catch (const std::invalid_argument& e)
	{
		fprintf(stderr, "%s\n", e.what());
		return false;
	}

	// convert to 2 ^ n
	if(n_objects_raise)
		NUM_OBJECTS = 1 << n_objects_raise;

	printf("Args used: ITERATIONS: %u; NUM_OBJECTS: %llu; NUM_BLOCKS: %u; NUM_THREADS: %u; SUB_STEPS: %u;\n\tMIN_RAD: %.2f; MAX_RAD: %.2f; CALCULATE_RAD: %s; MIN_X: %.2f; MIN_Y: %.2f; MAX_X: %.2f; MAX_Y: %.2f\n",
		ITERATIONS, NUM_OBJECTS, NUM_BLOCKS, NUM_THREADS, SUB_STEPS, MIN_RAD, MAX_RAD, CALCULATE_RAD ? "true" : "false", MIN_X, MIN_Y, MAX_X, MAX_Y);  // NOLINT(clang-diagnostic-double-promotion)

	return true;
}

int main(int argc, char** argv)
{
	// Create a solver instance
	fen::col_solver::CreateInstance();

	// Default values for parameters
	size_t NUM_OBJECTS = 1 << 18;

	unsigned int NUM_BLOCKS = 100;
	unsigned int NUM_THREADS = 512;
	unsigned int SUB_STEPS = 8;
	unsigned int ITERATIONS = 30;
	float MIN_RAD = 0.5f;
	float MAX_RAD = 1.0f;
	bool CALCULATE_RAD = false;

	float MIN_X = -2048.0f;
	float MIN_Y = -2048.0f;
	float MAX_X = +2048.0f;
	float MAX_Y = +2048.0f;

	// Read user args
	if(!read_args(argc, argv, NUM_OBJECTS, NUM_BLOCKS, NUM_THREADS, SUB_STEPS, ITERATIONS, MIN_RAD, MAX_RAD, CALCULATE_RAD, MIN_X, MIN_Y, MAX_X, MAX_Y))
	{
		print_usage(argv[0]);
		return 0;
	}

	// Create host vectors
	auto positions = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto delta_mov = thrust::device_vector<float>(NUM_OBJECTS * DIM);
	auto radius = thrust::device_vector<float>(NUM_OBJECTS);

	// Get random seed
	auto seed = time(nullptr);
	srand(seed);
	printf("Seed used: %llu\n", seed);

	// Initialize objects using the GPU
	fen::init_objects(seed, NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, MIN_X, MAX_X, MIN_Y, MAX_Y, MIN_RAD, MAX_RAD);

	// Print a couple entities 
	//thrust::host_vector<float> h_positions = positions;
	//thrust::host_vector<float> h_radius = radius;
	//print_entities(h_positions, h_radius, NUM_OBJECTS, 16);

	// Get a solver instance
	auto col_solver = fen::col_solver::Instance();

	// Reset the instance
	col_solver->reset(NUM_OBJECTS, CALCULATE_RAD ? -1.0f : MAX_RAD, MIN_X, MIN_Y, MAX_X, MAX_Y);

	// Run the solver ITERATIONS times 
	for (unsigned i = 0; i < ITERATIONS; ++i) {

		// Only get the collisions value for the first sub-step
		const unsigned int collisions_1 = col_solver->solve_cols(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);

		// Run the remaining sub-steps
		for (unsigned s = 0; s < SUB_STEPS - 1u; ++s)
		{
			col_solver->solve_cols(NUM_BLOCKS, NUM_THREADS, positions, radius, delta_mov, NUM_OBJECTS);
		}

		// Add a step to the profiler (to compute the avg times accordingly)
		col_solver->get_profiler().next_step();

		printf("col:%u\n", collisions_1);
	}

	// Print the profiling results
	std::cout << '\n';
	col_solver->get_profiler().print_avg_times<Solver_Execution_Steps>(std::cout);
	std::cout << '\n';
	col_solver->get_profiler().print_times(std::cout);

    return 0;
}
